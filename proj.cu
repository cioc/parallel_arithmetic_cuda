
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

/*
	TODO project: reciprocal 
*/

/*
	TODO project: negative number human readable
*/

//KEY CONSTANTS

//THIS IS THE STARTING SIZE FOR THE TWO TEMP BUFFERS 
//THE LARGER YOU EXPECT YOUR NUMBERS TO GROW, INCREASES THE SIZE OF THIS VALUE TO INCREASE PERFORMANCE
const unsigned int ORIGINAL_TEMP_BUFFER_SIZE = 1024;
//THESE TWO WILL SERVE AS OUR TEMPORARY BUFFERS IN OUR COMPUTATIONS
int *temp_buffer1;
int *temp_buffer2;
int *temp_buffer3;
unsigned int temp_buffer_size;

//NORMALIZATION KEY VALUES
const unsigned int BITS_PER_DIGIT = 32;
const unsigned int NORMALIZATION_EXPANSION = (unsigned int)ceil((BITS_PER_DIGIT * log(2.0)) / (log(10.0)));

//KEY PROCESSING CONSTANTS
//THIS IS THE DEVICE NUMBER THAT WE WILL DO OUR CALCULATIONS ON
const int DEVICE_NUM = 0;
int MAX_THREADS_PER_BLOCK;

//THE FOLLOWING IS HELPFUL INPUT CODE
//BCD - binary coded decimal
//A BCD IS THE DATA STRUCTURE THAT WE WILL USE TO REPRESENT OUR LARGE NUMBERS

//decpos IS THE POSITION OF THE DECIMAL IN THE NUMBER
//length IS THE NUMBER OF DIGITS IN THE NUMBER
//values IS AN ARRAY OF THE DIGITS
//gpuP IS THE POINTER TO THE DIGITS THAT HAVE BEEN COPIED TO THE GPU'S MEMORY
typedef struct bcd {
	unsigned int decpos;
	unsigned int length; 
	int *values; 
	int *gpuP;
} bdc;

//THIS TAKES A STRING REPRESENTATION OF OUR NUMBER, SUCH AS "123456544.23" AND LOADS IT INTO A BCD
void bcdFromString(char* input, bcd* output);
//THIS CREATES A BCD THAT CAN STORE A NUMBER WITH len DIGITS
bcd* createBcd(unsigned int len);
//THIS PRINTS A BCD OUT TO THE CONSOLE
void printBcd(bcd* input);
void printBcdNotNormal(bcd* input);
void zeroBcd(bcd* input);
void freeBcd(bcd* input);

//THE IMPLEMENTATION OF THESE THREE FUNCTIONS FOLLOWS:

void bcdFromString(char* input, bcd* output)
{
	unsigned int len = strlen(input);
	unsigned int lenstore = len;
	unsigned int x = 0;
	unsigned char decFound = 0;
	unsigned int negative = 0;
	for (x = 0; x < len; ++x)
	{
		char temp = input[x];
		switch (temp)
		{
			case '-':
				//lenstore -= 1;
				negative = 1;
			break;
			case '0':
				if (decFound > 0)
				{
					output->values[x - 1] = 0;
				}
				else
				{
					output->values[x] = 0;
				}
			break;
			case '1':
				if (decFound > 0)
				{
					output->values[x - 1] = 1;
				}
				else
				{
					output->values[x] = 1;
				}
			break;
			case '2':
				if (decFound > 0)
				{
					output->values[x - 1] = 2;
				}
				else
				{
					output->values[x] = 2;
				}
			break;
			case '3':
				if (decFound > 0)
				{
					output->values[x - 1] = 3;
				}
				else
				{
					output->values[x] = 3;
				}
			break;
			case '4':
				if (decFound > 0)
				{
					output->values[x - 1] = 4;
				}
				else
				{
					output->values[x] = 4;
				}
			break;
			case '5':
				if (decFound > 0)
				{
					output->values[x - 1] = 5;
				}
				else
				{
					output->values[x] = 5;
				}
			break;
			case '6':
				if (decFound > 0)
				{
					output->values[x - 1] = 6;
				}
				else
				{
					output->values[x] = 6;
				}
			break;
			case '7':
				if (decFound > 0)
				{
					output->values[x - 1] = 7;
				}
				else
				{
					output->values[x] = 7;
				}
			break;
			case '8':
				if (decFound > 0)
				{
					output->values[x - 1] = 8;
				}
				else
				{
					output->values[x] = 8;
				}
			break;
			case '9':
				if (decFound > 0)
				{
					output->values[x - 1] = 9;
				}
				else
				{
					output->values[x] = 9;
				}
			break;
			case '.':
				output->decpos = x;
				lenstore -= 1;
				decFound = 1;
			break;
		}
	}
	output->length = lenstore;
	if (negative == 1)
	{
		int i = 0;
		for(i = 0; i < lenstore; i++)
		{
			output->values[i] = output->values[i] * (-1);
		}
	}
	if (decFound == 0)
	{
		output->decpos = lenstore;
	}
}

bcd* createBcd(unsigned int len)
{
	bcd* output = (bcd *)malloc(sizeof(bcd));
	output->length = len;
	output->values = (int *)malloc(len * sizeof(int));
	return output;
}

void zeroBcd(bcd* input)
{
	int c = 0;
	for (c = 0; c < input->length; ++c)
	{
		*(input->values + c) = 0; 
	}
}

void printBcd(bcd* input)
{
	int i = 0;
	for(i = 0; i < input->length; i++)
	{
		if (i == input->decpos)
		{
			printf(".");
		}
		printf("%i", input->values[i]);
	}
	printf("\n");
}

void printBcdNotNormal(bcd* input)
{
	int i = 0;
	for(i = 0; i < input->length; i++)
	{
		if (i == input->decpos)
		{
			printf(".");
		}
		printf("%i", input->values[i]);
		printf("|");
	}
	printf("\n");
}

void freeBcd(bcd* input)
{
	hipFree(input->gpuP);
	free(input->values);
	free(input);
}
//cudaFree

//GPU CODE

//THIS FUNCTION LOADS THE VALUES OF A BCD INTO TEH GPU'S MEMORY AND SETS THE gpuP OF THE BCD TO POINT TO THE GPU-STORED VALUES
void loadBcdIntoGPU(bcd* input);
//THIS COPIES BACK THE RESULTS FROM THE GPU TO THE BCD
void getCompResult(bcd* output);

void loadBcdIntoGPU(bcd* input)
{
	hipMalloc(&input->gpuP, input->length * sizeof(int));
	hipMemcpy(input->gpuP, input->values, input->length * sizeof(int), hipMemcpyHostToDevice);
}
void getCompResult(bcd* output)
{
	hipMemcpy(output->values,output->gpuP, output->length * sizeof(int), hipMemcpyDeviceToHost);
}

//THE FOLLOWING IS ALL SETUP CODE

void cudaSetup();
void initTempBuffers();
void reallocTempBuffers(unsigned int size);
void freeTempBuffers();
void zeroTempBuffers();

//THIS IS THE MAIN SETUP FUNCTION.  CALL THIS EARLY ON IN MAIN.  BEFORE ANY ADDITIONS OR MULTIPLICATIONS ON BCD'S
//will call initTempBuffers
void cudaSetup()
{
	//LET'S FIGURE OUT THE MAXIMUM THREADS PER BLOCK
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, DEVICE_NUM);
	MAX_THREADS_PER_BLOCK = deviceProp.maxThreadsPerBlock;
	initTempBuffers();
}


void initTempBuffers()
{
	hipMalloc(&temp_buffer1, ORIGINAL_TEMP_BUFFER_SIZE * sizeof(int));
	hipMalloc(&temp_buffer2, ORIGINAL_TEMP_BUFFER_SIZE * sizeof(int));
	hipMalloc(&temp_buffer3, ORIGINAL_TEMP_BUFFER_SIZE * sizeof(int));
	hipMemset(temp_buffer1, 0, ORIGINAL_TEMP_BUFFER_SIZE * sizeof(int));
	hipMemset(temp_buffer2, 0, ORIGINAL_TEMP_BUFFER_SIZE * sizeof(int));
	hipMemset(temp_buffer3, 0, ORIGINAL_TEMP_BUFFER_SIZE * sizeof(int));	
	temp_buffer_size = ORIGINAL_TEMP_BUFFER_SIZE;
}

void reallocTempBuffers(unsigned int size)
{
	if (temp_buffer_size < size)
	{
		freeTempBuffers();
		hipMalloc(&temp_buffer1, size * sizeof(int));
		hipMalloc(&temp_buffer2, size * sizeof(int));
		hipMalloc(&temp_buffer3, size * sizeof(int));
		hipMemset(temp_buffer1, 0, size * sizeof(int));
		hipMemset(temp_buffer2, 0, size * sizeof(int));
		hipMemset(temp_buffer3, 0, size * sizeof(int));
		temp_buffer_size = size;
	}
}

void zeroTempBuffers()
{
	hipMemset(temp_buffer1, 0, temp_buffer_size * sizeof(int));
	hipMemset(temp_buffer2, 0, temp_buffer_size * sizeof(int));
	hipMemset(temp_buffer3, 0, temp_buffer_size * sizeof(int));
}

void freeTempBuffers()
{
	hipFree(temp_buffer1);
	hipFree(temp_buffer2);
	hipFree(temp_buffer3);
}

//MEMORY REQUIREMENT CALCULATION CODE

unsigned int memReqForAddition(bcd* num1, bcd* num2);
unsigned int memReqForMulitiplcation(bcd* num1, bcd* num2);

unsigned int memReqForAddition(bcd* num1, bcd* num2)
{
	unsigned int maxlen = 0; 
	if (num1->length > num2->length)
	{
		maxlen = num1->length;
	}
	else
	{
		maxlen = num2->length;
	}
	return (maxlen + NORMALIZATION_EXPANSION + 1);
}

unsigned int memReqForMulitiplcation(bcd* num1, bcd* num2)
{
	return num1->length + num2->length + (2 * NORMALIZATION_EXPANSION);
}

//DECIMAL POSITION CHANGE CODE

unsigned int decimalMovementAddition(bcd* num1, bcd* num2, unsigned int memReq);
unsigned int decimalMovementMultiplication(bcd* num1, bcd* num2, unsigned int memReq);

unsigned int decimalMovementAddition(bcd* num1, bcd* num2, unsigned int memReq)
{
	if (num1->length > num2->length)
	{
		return (memReq - num1->length) + num1->decpos;
	}
	else
	{
		return (memReq - num2->length) + num2->decpos;
	}
}

/*
	TODO START HERE
*/
unsigned int decimalMovementMultiplication(bcd* num1, bcd* num2, unsigned int memReq)
{
	return memReq - ((num1->length - num1->decpos) + (num2->length - num2->decpos));
}

//KERNELS

__global__ void addition(int *num1, int *num2, unsigned int num1Len, unsigned int num2Len, unsigned int num1offset, unsigned int num2offset, int *temp_buffer1, int *temp_buffer2, int *output, unsigned int memReq, unsigned int reps);
__global__ void normalize(int *num,unsigned int num1Len, int *result,unsigned int memReq);
__global__ void multiplication(int *num1, int *num2, unsigned int num1Len, unsigned int num2Len, unsigned int num1offset, unsigned int num2offset, int *temp_buffer1, int *temp_buffer2, int *temp_buffer3, int *output, unsigned int memReq, unsigned int reps);

/*
	TODO normalize: need to get this working for reps.  EG: numbers longer the 512 digits
*/
/*
	TODO normalize: need to make this work for negative numbers
*/

__global__ void normalize(int *num, unsigned int numLen, int *result,unsigned int memReq, unsigned int reps)
{
	int x = threadIdx.x;
	if (x < memReq)
	{
		if (reps == 1)
		{
			if (x >= memReq - numLen)
			{
				result[x] = num[x - (memReq - numLen)];
			}
		}
		else
		{
			//result[x] = num[x - (memReq - numLen)];
			int d = 0;
			for (d = 0; d < reps; ++d)
			{
				if (d == 0)
				{
					if (x >= memReq - numLen)
					{
						result[x + (512 * d)] = num[x + (512 * d) - (memReq - numLen)];
					}
				}
				else
				{
					if ((x + (512 * d) - (memReq - numLen)) < numLen)
					result[x + (512 * d)] = num[x + (512 * d) - (memReq - numLen)];
				}
			}
		}
	}
	
	__shared__ int carry;
	carry = 1;
	__syncthreads();
	while (carry)
	{
		if (reps == 1)
		{
			int c = 0;
			if (x < memReq)
			{
				c = result[x] / 10;
				result[x] %= 10;
			}
			__syncthreads();
			if (x < memReq && x != 0)
			{
				result[x - 1] += c;
			}
		}
		else
		{
			int d = 0;
			for(d = 0; d < reps; ++d)
			{
				int c = 0;
				if ((x + (512 * d)) < memReq)
				{
					c = result[x + (512 * d)] / 10;
					result[x + (512 * d)] %= 10;
				}
				__syncthreads();
				if ((x + (512 * d)) < memReq && (x != 0))
				{
					result[x - 1 + (512 * d)] += c;
				}
			}
		}
		carry = 0;
		__syncthreads();
		if (x < memReq)
		{
			if (reps == 1)
			{
				if (abs(result[x]) > 9)
				{
					carry = 1;
				}
			}
			else
			{
				int d = 0;
				for(d = 0; d < reps; ++d)
				{
					if ((x + (512 * d)) < memReq && (abs(result[x + (512 * d)]) > 9))
					{
						carry = 1;
					}
				}
			}
		}
		__syncthreads();
	}
}

/*
	TODO addition kernel: need to add in normalization
*/
__global__ void addition(int *num1, int *num2, unsigned int num1Len, unsigned int num2Len, unsigned int num1offset, unsigned int num2offset, int *temp_buffer1, int *temp_buffer2, int *output, unsigned int memReq, unsigned int reps)
{
	int x = threadIdx.x;
	if (reps == 1)
	{
		if (x < memReq)
		{
			if (x >= memReq - num1Len)
			{
				temp_buffer1[x] = num1[x - (memReq - num1Len)];
			}
			if (x >= memReq - num2Len)
			{
				temp_buffer2[x] = num2[x - (memReq - num2Len)];
			}
		}
	}
	else
	{
		int d = 0;
		for (d = 0; d < reps; ++d)
		{
			if (d == 0)
			{
				if (x >= memReq - num1Len)
				{
					temp_buffer1[x + (512 * d)] = num1[x + (512 * d) - (memReq - num1Len)];
				}
				if (x >= memReq - num2Len)
				{
					temp_buffer2[x + (512 * d)] = num2[x + (512 * d) - (memReq - num2Len)];
				}
			}
			else
			{
				if ((x + (512 * d) - (memReq - num1Len)) < num1Len)
				{
					temp_buffer1[x + (512 * d)] = num1[x + (512 * d) - (memReq - num1Len)];
				}
				if ((x + (512 * d) - (memReq - num2Len)) < num2Len)
				{
					temp_buffer2[x + (512 * d)] = num2[x + (512 * d) - (memReq - num2Len)];
				}
			}
		}
	}
	//move everything to temp buffers
	__shared__ int carry;
	carry = 0;
	__syncthreads();
	if (reps == 1)
	{
		if (((unsigned int)(temp_buffer1[x] & ((unsigned int)3 << 30)) > 0) || ((unsigned int)(temp_buffer2[x] & ((unsigned int)3 << 30)) > 0))
		{
			carry = 1;
		}
	}
	else
	{
		int d = 0;
		for (d = 0; d < reps; ++d)
		{
			if ((x + (512 * d)) < memReq)
			{
				if (((unsigned int)(temp_buffer1[x + (512 * d)] & ((unsigned int)3 << 30)) > 0) || ((unsigned int)(temp_buffer2[x + (512 * d)] & ((unsigned int)3 << 30)) > 0))
				{
					carry = 1;
				}
			}
		}
	}
	__syncthreads();
	while (carry)
	{
		if (reps == 1)
		{
			int c1 = 0;
			int c2 = 0;
			if (x < memReq)
			{
				c1 = temp_buffer1[x] / 10;
				c2 = temp_buffer2[x] / 10;
				temp_buffer1[x] %= 10;
				temp_buffer2[x] %= 10;
			}
			__syncthreads();
			if (x < memReq && x != 0)
			{
				temp_buffer1[x - 1] += c1;
				temp_buffer2[x - 1] += c2;
			}
			carry = 0;
			__syncthreads();
			if (x < memReq)
			{
				if ((abs(temp_buffer1[x]) > 9) || (abs(temp_buffer2[x]) > 9))
				{
					carry = 1;
				}
			}
			__syncthreads();
		}
		else
		{
			int d = 0;
			for(d = 0; d < reps; ++d)
			{
				int c1 = 0;
				int c2 = 0;
				if ((x + (512 * d)) < memReq)
				{
					c1 = temp_buffer1[x + (512 * d)] / 10;
					c2 = temp_buffer2[x + (512 * d)] / 10;
					temp_buffer1[x + (512 * d)] %= 10;
					temp_buffer2[x + (512 * d)] %= 10;
				}
				__syncthreads();
				if ((x + (512 * d)) < memReq && (x != 0))
				{
					temp_buffer1[x - 1 + (512 * d)] += c1;
					temp_buffer2[x - 1 + (512 * d)] += c2;
				}
			}
		}
		carry = 0;
		__syncthreads();
		if (x < memReq)
		{
			if (reps == 1)
			{
				if ((abs(temp_buffer1[x]) > 9) || (abs(temp_buffer2[x]) > 9))
				{
					carry = 1;
				}
			}
			else
			{
				int d = 0;
				for(d = 0; d < reps; ++d)
				{
					if ((x + (512 * d)) < memReq && ((abs(temp_buffer1[x + (512 * d)]) > 9) || (abs(temp_buffer2[x + (512 * d)]) > 9)))
					{
						carry = 1;
					}
				}
			}
		}
		__syncthreads();
	}
	if (x < memReq)
	{
		if (reps == 1)
		{
			if (((x + num1offset) < memReq) && ((x + num2offset) < memReq))
			{
				output[x] = temp_buffer1[x + num1offset] + temp_buffer2[x + num2offset];
			}
			else if ((x + num2offset) < memReq)
			{
				output[x] = temp_buffer2[x + num2offset];
			}
			else if ((x + num1offset) < memReq)
			{
				output[x] = temp_buffer1[x + num1offset];
			}
			else
			{
				//do nothing 
			}
		}
		else
		{
			int d = 0;
			for(d = 0; d < reps; ++d)
			{
				if ((((x + (512 * d)) + num1offset) < memReq) && (((x + (512 * d)) + num2offset) < memReq))
				{
					output[(x + (512 * d))] = temp_buffer1[(x + (512 * d)) + num1offset] + temp_buffer2[(x + (512 * d)) + num2offset];
				}
				else if (((x + (512 * d)) + num2offset) < memReq)
				{
					output[(x + (512 * d))] = temp_buffer2[(x + (512 * d)) + num2offset];
				}
				else if (((x + (512 * d)) + num1offset) < memReq)
				{
					output[(x + (512 * d))] = temp_buffer1[(x + (512 * d)) + num1offset];
				}
				else
				{
					//do nothing 
				}
			}
		}
	}		
}

//multiplication<<<1,MAX_THREADS_PER_BLOCK>>>(num1->gpuP, num2->gpuP, num1->length, num2->length, dec1_offset, dec2_offset, temp_buffer1, temp_buffer2, temp_buffer3, output->gpuP, result_req, reps);

__global__ void multiplication(int *num1, int *num2, unsigned int num1Len, unsigned int num2Len, unsigned int num1offset, unsigned int num2offset, int *temp_buffer1, int *temp_buffer2, int *temp_buffer3, int *output2, unsigned int memReq, unsigned int reps)
{
	int x = threadIdx.x;
	if (reps == 1)
	{
		if (x < memReq)
		{
			if (x >= memReq - num1Len)
			{
				temp_buffer1[x] = num1[x - (memReq - num1Len)];
			}
			if (x >= memReq - num2Len)
			{
				temp_buffer2[x] = num2[x - (memReq - num2Len)];
			}
		}
	}
	else
	{
		int d = 0;
		for (d = 0; d < reps; ++d)
		{
			if (d == 0)
			{
				if (x >= memReq - num1Len)
				{
					temp_buffer1[x + (512 * d)] = num1[x + (512 * d) - (memReq - num1Len)];
					//output2[x + (512 * d)] = num1[x + (512 * d) - (memReq - num1Len)];
				}
				if (x >= memReq - num2Len)
				{
					temp_buffer2[x + (512 * d)] = num2[x + (512 * d) - (memReq - num2Len)];
				}
			}
			else
			{
				if ((x + (512 * d) - (memReq - num1Len)) < num1Len)
				{
					temp_buffer1[x + (512 * d)] = num1[x + (512 * d) - (memReq - num1Len)];
					//output2[x + (512 * d)] = num1[x + (512 * d) - (memReq - num1Len)];
					
				}
				if ((x + (512 * d) - (memReq - num2Len)) < num2Len)
				{
					temp_buffer2[x + (512 * d)] = num2[x + (512 * d) - (memReq - num2Len)];
				}
			}
		}
	}
	//move everything to temp buffers
	__shared__ int carry;
	carry = 1;
	__syncthreads();
	while (carry)
	{
		if (reps == 1)
		{
			
			int c1 = 0;
			int c2 = 0;
			if (x < memReq)
			{
				c1 = temp_buffer1[x] / 10;
				c2 = temp_buffer2[x] / 10;
				temp_buffer1[x] %= 10;
				temp_buffer2[x] %= 10;
			}
			__syncthreads();
			if (x < memReq && x != 0)
			{
				temp_buffer1[x - 1] += c1;
				temp_buffer2[x - 1] += c2;
			}
			
		}
		else
		{
			
			int d = 0;
			for(d = 0; d < reps; ++d)
			{
				int c1 = 0;
				int c2 = 0;
				if ((x + (512 * d)) < memReq)
				{
					c1 = temp_buffer1[x + (512 * d)] / 10;
					c2 = temp_buffer2[x + (512 * d)] / 10;
					temp_buffer1[x + (512 * d)] %= 10;
					temp_buffer2[x + (512 * d)] %= 10;
				}
				__syncthreads();
				if ((x + (512 * d)) < memReq && (x != 0))
				{
					temp_buffer1[x - 1 + (512 * d)] += c1;
					temp_buffer2[x - 1 + (512 * d)] += c2;
				}
			}
			
		}
		carry = 0;
		__syncthreads();
		if (reps == 1)
		{
			if ((x < memReq) && ((abs(temp_buffer1[x]) > 9) || (abs(temp_buffer2[x]) > 9)))
			{
				carry = 1;
			}
		}
		else
		{
			int d = 0;
			for(d = 0; d < reps; ++d)
			{
				if ((x + (512 * d)) < memReq && ((abs(temp_buffer1[x + (512 * d)]) > 9) || (abs(temp_buffer2[x + (512 * d)]) > 9)))
				{
					carry = 1;
				}
			}
		}
		__syncthreads();
	}
	
	//good till here
	//TEST INTITIAL NORMALIZATION 
	
	__shared__ int multCount;
	multCount = 0;
	__syncthreads();
	//output2[x] = reps;
	
	//__syncthreads(); <-- uncomment this too
	//TEST JUST ONE ITERATION
	//while (multCount < num2Len)
	
	while (multCount < num2Len)
	{
		int tempMultCountStore = multCount;
		tempMultCountStore += 1;
		
		if (reps == 1)
		{
			if (x < memReq)
			{
				if (x > multCount)
				{
					temp_buffer3[x - multCount] = temp_buffer2[memReq - multCount - 1] * temp_buffer1[x];
				}
			}
			//check for overflow
		}
		else
		{
			int d = 0;
			for(d = 0; d < reps; ++d)
			{
				if (d == 0)
				{
					if ((x > multCount) && ((x + (512 * d)) < memReq))
					{
						temp_buffer3[(x + (512 * d)) - multCount] = temp_buffer2[memReq - multCount - 1] * temp_buffer1[(x + (512 * d))];
					}
				}
				else
				{
					if ((x + (512 * d)) < memReq)
					{
						temp_buffer3[(x + (512 * d)) - multCount] = temp_buffer2[memReq - multCount - 1] * temp_buffer1[(x + (512 * d))];
					}
				}
			}
		}
		
		carry = 0;
		__syncthreads();
		

		int d = 0;
		if (reps == 1)
		{
			for (d = 0; d <= reps; ++d)
			{
				if ((x + (512 * d)) < memReq)
				{
					if (((unsigned int)(temp_buffer3[x + (512 * d)] & ((unsigned int)3 << 30)) > 0) || ((unsigned int)(output2[x + (512 * d)] & ((unsigned int)3 << 30)) > 0))
					{
						carry = 1;
					}
				}
			}
		}
		else
		{
			for (d = 0; d < reps; ++d)
			{
				if ((x + (512 * d)) < memReq)
				{
					if (((unsigned int)(temp_buffer3[x + (512 * d)] & ((unsigned int)3 << 30)) > 0) || ((unsigned int)(output2[x + (512 * d)] & ((unsigned int)3 << 30)) > 0))
					{
						carry = 1;
					}
				}
			}
		}
		__syncthreads();
		
		while (carry)
		{
			if (reps == 1)
			{
				int c1 = 0;
				int c2 = 0;
				if ((reps == 1) && (x < memReq))
				{
					c1 = temp_buffer3[x] / 10;
					c2 = output2[x] / 10;
					temp_buffer3[x] %= 10;
					output2[x] %= 10;
				}
				__syncthreads();
				if (x < memReq && x != 0)
				{
					temp_buffer3[x - 1] += c1;
					output2[x - 1] += c2;
				}
				carry = 0;
				__syncthreads();
				if (x < memReq)
				{
					if ((abs(temp_buffer3[x]) > 9) || abs((output2[x]) > 9))
					{
						carry = 1;
					}
				}
				__syncthreads();
			}
			else
			{
				
				int d = 0;
				for(d = 0; d < reps; ++d)
				{
					int c1 = 0;
					int c2 = 0;
					if ((x + (512 * d)) < memReq)
					{
						c1 = temp_buffer3[x + (512 * d)] / 10;
						c2 = output2[x + (512 * d)] / 10;
						temp_buffer3[x + (512 * d)] %= 10;
						output2[x + (512 * d)] %= 10;
					}
					__syncthreads();
					if (d == 0)
					{
						if ((x != 0) && ((x + (512 * d)) < memReq))
						{
							//SOMEHOW DESYNCRONIZED
							temp_buffer3[x - 1 + (512 * d)] += c1;
							output2[x - 1 + (512 * d)] += c2;
						}
					}
					else
					{
						if (((x + (512 * d)) < memReq))
						{
							//SOMEHOW DESYNCRONIZED
							temp_buffer3[x - 1 + (512 * d)] += c1;
							output2[x - 1 + (512 * d)] += c2;
						}
					}
					__syncthreads();
					carry = 0;
					int d = 0;
					for(d = 0; d < reps; ++d)
					{
						if ((x + (512 * d)) < memReq && ((abs(temp_buffer3[x + (512 * d)]) > 9) || (abs(output2[x + (512 * d)]) > 9)))
						{
							carry = 1;
						}
					}
					__syncthreads();
				}
				
			}
		}
		//perform addition 
		if (reps == 1)
		{
			if (x < memReq)
			{
				output2[x] += temp_buffer3[x];
				temp_buffer3[x] = 0;
			}
			//check for overflow
		}
		else
		{
			int d = 0;
			for(d = 0; d < reps; ++d)
			{
				if ((x + (512 * d)) < memReq)
				{
					output2[x + (512 * d)] += temp_buffer3[x + (512 * d)];
					temp_buffer3[x + (512 * d)] = 0;
				}
			}
		}
		//update counter
		multCount = tempMultCountStore;
		__syncthreads();	
	}
}


//ARITHMETIC FUNCTIONS

bcd* normalize(bcd *num)
{
	unsigned int memReq = NORMALIZATION_EXPANSION + num->length;
	bcd *output = createBcd(memReq);
	zeroBcd(output);
	loadBcdIntoGPU(output);
		
	output->decpos = num->decpos + (memReq - num->length);
	unsigned int reps = (memReq / MAX_THREADS_PER_BLOCK) + 1;
	//printf("reps: %u",reps);
	normalize<<<1,MAX_THREADS_PER_BLOCK>>>(num->gpuP, num->length, output->gpuP,memReq,reps);
	
	return output;
}

bcd* add(bcd *num1, bcd *num2)
{
	//first calc memory requirement for result
	unsigned int result_req = memReqForAddition(num1, num2);
	bcd *output = createBcd(result_req);
	
	if (result_req > temp_buffer_size)
	{
		reallocTempBuffers(result_req * 2);
	}
	
	zeroBcd(output);
	loadBcdIntoGPU(output);
	output->decpos = decimalMovementAddition(num1, num2, result_req);
	unsigned int reps = (result_req / MAX_THREADS_PER_BLOCK) + 1;
	
	//now we'll figure out the decimal offset
	unsigned int decdiff1 = num1->length - num1->decpos;
	unsigned int decdiff2 = num2->length - num2->decpos;
	unsigned int dec1_offset = 0;
	unsigned int dec2_offset = 0;
	
	if (decdiff1 > decdiff2)
	{
		dec2_offset = decdiff1 - decdiff2;
	}
	else
	{
		dec1_offset = decdiff2 - decdiff1;
	}
	zeroTempBuffers();
	addition<<<1,MAX_THREADS_PER_BLOCK>>>(num1->gpuP, num2->gpuP, num1->length, num2->length, dec1_offset, dec2_offset, temp_buffer1, temp_buffer2, output->gpuP, result_req, reps);
	
	return output;
}

bcd *multiply(bcd *num1, bcd *num2)
{
	unsigned int result_req = memReqForMulitiplcation(num1, num2);
	bcd *output = createBcd(result_req);
	
	if (result_req > temp_buffer_size)
	{
		reallocTempBuffers(result_req * 2);
	}
	zeroBcd(output);
	loadBcdIntoGPU(output);
	output->decpos = decimalMovementMultiplication(num1, num2, result_req);
	unsigned int reps = (result_req / MAX_THREADS_PER_BLOCK) + 1;
	//printf("REPS: %u\n", reps);
	
	//now we'll figure out the decimal offset
	unsigned int decdiff1 = num1->length - num1->decpos;
	unsigned int decdiff2 = num2->length - num2->decpos;
	unsigned int dec1_offset = 0;
	unsigned int dec2_offset = 0;
	
	if (decdiff1 > decdiff2)
	{
		dec2_offset = decdiff1 - decdiff2;
	}
	else
	{
		dec1_offset = decdiff2 - decdiff1;
	}
	//printf("RESULT REQ: %u\n", result_req);
	zeroTempBuffers();
	multiplication<<<1,MAX_THREADS_PER_BLOCK>>>(num1->gpuP, num2->gpuP, num1->length, num2->length, dec1_offset, dec2_offset, temp_buffer1, temp_buffer2, temp_buffer3, output->gpuP, result_req, reps);
	
	return output; 
}

//THIS IS A TESTBED FOR OUR LIBRARY 
//AN EXAMPLE
int main()
{
	//bcd *num1 = createBcd(903);
	bcd *num1 = createBcd(7);
	bcd *num2 = createBcd(2);
	
	//bcdFromString("111111111111111111111111111111111111111111111112111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111121111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111111.9", num1);
	bcdFromString("112341.9", num1);
	bcdFromString("1.2", num2);
	printBcdNotNormal(num1);
	printBcd(num2);
	
	cudaSetup();
	loadBcdIntoGPU(num1);
	loadBcdIntoGPU(num2);
	bcd* result = multiply(num1, num2); 
	bcd* normResult = normalize(result);
	
	getCompResult(normResult);
	printf("\n");
	printBcd(normResult);
	//printBcd(result);
	
	freeBcd(num1);
	freeBcd(num2);
	freeBcd(result);
	freeBcd(normResult);
	freeTempBuffers();
	return 0;
}